
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_deconv_layer.hpp"

namespace caffe {

__global__ void sync_deconv_groups() {}

template <typename Dtype>
void CuDNNDeconvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

#if CUDNN_VERSION_MIN(7,0,0)
    if (multiple_handles_)
      {
#endif

        // Forward through cuDNN in parallel over groups.
        for (int g = 0; g < this->group_; g++) {
          // Filters.
          CUDNN_CHECK(cudnnConvolutionBackwardData(
                                                   handle_[g],
                                                   cudnn::dataType<Dtype>::one,
                                                   filter_desc_,
                                                   weight + this->weight_offset_ * g,
                                                   bottom_descs_[i],
                                                   bottom_data + bottom_offset_ * g,
                                                   conv_descs_[i],
                                                   bwd_data_algo_[i],
                                                   workspace[g],
                                                   workspace_bwd_data_sizes_[i],
                                                   cudnn::dataType<Dtype>::zero,
                                                   top_descs_[i],
                                                   top_data + top_offset_ * g));

          // Bias.
          if (this->bias_term_) {
            const Dtype* bias_data = this->blobs_[1]->gpu_data();
            CUDNN_CHECK(cudnnAddTensor(handle_[g],
                                       cudnn::dataType<Dtype>::one,
                                       bias_desc_,
                                       bias_data + bias_offset_ * g,
                                       cudnn::dataType<Dtype>::one,
                                       top_descs_[i],
                                       top_data + top_offset_ * g));
          }
        }

        // Synchronize the work across groups, each of which went into its own
        // stream, by launching an empty kernel into the default (null) stream.
        // NOLINT_NEXT_LINE(whitespace/operators)
        sync_deconv_groups<<<1, 1>>>();
#if CUDNN_VERSION_MIN(7,0,0)
      }
    else
      {
        CUDNN_CHECK(cudnnConvolutionBackwardData(
                                                 handle_[0],
                                                 cudnn::dataType<Dtype>::one,
                                                 filter_desc_,
                                                 weight,
                                                 bottom_descs_[i],
                                                 bottom_data,
                                                 conv_descs_[i],
                                                 bwdDataPerf_[i].algo,
                                                 workspaceData,
                                                 bwdDataPerf_[i].memory,
                                                 cudnn::dataType<Dtype>::zero,
                                                 top_descs_[i],
                                                 top_data));

        // Bias.
        if (this->bias_term_) {
          const Dtype* bias_data = this->blobs_[1]->gpu_data();
          CUDNN_CHECK(cudnnAddTensor(handle_[0],
                                     cudnn::dataType<Dtype>::one,
                                     bias_desc_,
                                     bias_data,
                                     cudnn::dataType<Dtype>::one,
                                     top_descs_[i],
                                     top_data));
        }
      }
#endif
  }
}

template <typename Dtype>
void CuDNNDeconvolutionLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.


#if CUDNN_VERSION_MIN(7,0,0)
    if (multiple_handles_)
      {
#endif

        for (int g = 0; g < this->group_; g++) {
          // Gradient w.r.t. bias.
          if (this->bias_term_ && this->param_propagate_down_[1]) {
            CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0 * this->group_ + g],
                                                     cudnn::dataType<Dtype>::one,
                                                     top_descs_[i],
                                                     top_diff + top_offset_ * g,
                                                     cudnn::dataType<Dtype>::one,
                                                     bias_desc_,
                                                     bias_diff + bias_offset_ * g));
          }

          // Gradient w.r.t. weights.
          if (this->param_propagate_down_[0]) {
            const Dtype* bottom_data = bottom[i]->gpu_data();
            CUDNN_CHECK(cudnnConvolutionBackwardFilter(
                                                       handle_[1 * this->group_ + g],
                                                       cudnn::dataType<Dtype>::one,
                                                       top_descs_[i],
                                                       top_diff + top_offset_ * g,
                                                       bottom_descs_[i],
                                                       bottom_data + bottom_offset_ * g,
                                                       conv_descs_[i],
                                                       bwd_filter_algo_[i],
                                                       workspace[1 * this->group_ + g],
                                                       workspace_bwd_filter_sizes_[i],
                                                       cudnn::dataType<Dtype>::one,
                                                       filter_desc_,
                                                       weight_diff + this->weight_offset_ * g));
          }

          // Gradient w.r.t. bottom data.
          if (propagate_down[i]) {
            if (weight == NULL) {
              weight = this->blobs_[0]->gpu_data();
            }
            Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
            CUDNN_CHECK(
                        cudnnConvolutionForward(handle_[2 * this->group_ + g],
                                                cudnn::dataType<Dtype>::one,
                                                top_descs_[i],
                                                top_diff + top_offset_ * g,
                                                filter_desc_,
                                                weight + this->weight_offset_ * g,
                                                conv_descs_[i],
                                                fwd_algo_[i],
                                                workspace[2 * this->group_ + g],
                                                workspace_fwd_sizes_[i],
                                                cudnn::dataType<Dtype>::zero,
                                                bottom_descs_[i],
                                                bottom_diff + bottom_offset_ * g));
          }
        }

        // Synchronize the work across groups, each of which went into its own
        // stream, by launching an empty kernel into the default (null) stream.
        // NOLINT_NEXT_LINE(whitespace/operators)
        sync_deconv_groups<<<1, 1>>>();
#if CUDNN_VERSION_MIN(7,0,0)
      }
    else
      {
        if (this->bias_term_ && this->param_propagate_down_[1]) {
          CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0],
                                                   cudnn::dataType<Dtype>::one,
                                                   top_descs_[i],
                                                   top_diff,
                                                   cudnn::dataType<Dtype>::one,
                                                   bias_desc_,
                                                   bias_diff));
      }

        // Gradient w.r.t. weights.
        if (this->param_propagate_down_[0]) {
          const Dtype* bottom_data = bottom[i]->gpu_data();
          CUDNN_CHECK(cudnnConvolutionBackwardFilter(
                                                     handle_[0],
                                                     cudnn::dataType<Dtype>::one,
                                                     top_descs_[i],
                                                     top_diff,
                                                     bottom_descs_[i],
                                                     bottom_data,
                                                     conv_descs_[i],
                                                     bwdFilterPerf_[i].algo,
                                                     workspaceData,
                                                     bwdFilterPerf_[i].memory,
                                                     cudnn::dataType<Dtype>::one,
                                                     filter_desc_,
                                                     weight_diff));
          }

          // Gradient w.r.t. bottom data.
        if (propagate_down[i]) {
          if (weight == NULL) {
            weight = this->blobs_[0]->gpu_data();
          }
          Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
          CUDNN_CHECK(cudnnConvolutionForward(handle_[0],
                                              cudnn::dataType<Dtype>::one,
                                              top_descs_[i],
                                              top_diff,
                                              filter_desc_,
                                              weight,
                                              conv_descs_[i],
                                              fwdPerf_[i].algo,
                                              workspaceData,
                                              fwdPerf_[i].memory,
                                              cudnn::dataType<Dtype>::zero,
                                              bottom_descs_[i],
                                              bottom_diff));
        }
      }
#endif
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNDeconvolutionLayer);

}  // namespace caffe
#endif
